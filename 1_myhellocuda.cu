#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void hello_world(){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("block id = [ %d ], thread id = [ %d ] hello cuda\n", blockIdx.x, idx);
}

int main() {
    hello_world<<<2,2>>>();
    hipDeviceSynchronize();
    return 0;
}