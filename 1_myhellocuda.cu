#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void hello_world(){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("block id = [ %d ], thread id = [ %d ] hello cuda\n", blockIdx.x, idx);
}

int main() {
    hello_world<<<2,2>>>();
    hipDeviceSynchronize(); // 用于同步cpu，实测关闭后不会输出
    return 0;
}